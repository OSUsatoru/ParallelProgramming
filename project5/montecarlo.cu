#include "hip/hip_runtime.h"
#include <stdio.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

// these two #defines are just to label things
// other than that, they do nothing:
#define IN
#define OUT

// print debugging messages?
#ifndef DEBUG
#define DEBUG	false
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE       16
#endif

// setting the number of trials in the monte carlo simulation:
#ifndef NUMTRIALS
#define NUMTRIALS	50000
#endif

// ranges for the random numbers:
const float GMIN =	20.0;	// ground distance in meters
const float GMAX =	30.0;	// ground distance in meters
const float HMIN =	10.0;	// cliff height in meters
const float HMAX =	40.0;	// cliff height in meters
const float DMIN  =	10.0;	// distance to castle in meters
const float DMAX  =	20.0;	// distance to castle in meters
const float VMIN  =	30.0;	// intial cnnonball velocity in meters / sec
const float VMAX  =	50.0;	// intial cnnonball velocity in meters / sec
const float THMIN = 	70.0;	// cannonball launch angle in degrees
const float THMAX =	80.0;	// cannonball launch angle in degrees

const float GRAVITY =	-9.8;	// acceleraion due to gravity in meters / sec^2
const float TOL = 5.0;		// tolerance in cannonball hitting the castle in meters
				// castle is destroyed if cannonball lands between d-TOL and d+TOL

// function prototypes:
float		Ranf( float, float );
int		Ranf( int, int );
void		TimeOfDaySeed( );
void CudaCheckError();

// degrees-to-radians -- callable from the device:
__device__
float
Radians( float d )
{
        return (M_PI/180.f) * d;
}

// the kernel:
__global__
void
MonteCarlo( float *dvs, float *dths, float *dgs, float *dhs, float *dds, int *dhits )
{
        unsigned int gid      = blockIdx.x*blockDim.x + threadIdx.x;

        // randomize everything:
        float v   = dvs[gid];
        float thr = Radians( dths[gid] );
        float vx  = v * cos(thr);
        float vy  = v * sin(thr);
        float  g  =  dgs[gid];
        float  h  =  dhs[gid];
        float  d  =  dds[gid];

        int numHits = 0;

        // see if the ball doesn't even reach the cliff:
        float t = -vy / ( 0.5*GRAVITY );
        float x = vx * t;
        if( x > g )
        {
            // see if the ball hits the vertical cliff face:
            t = g/vx;
            float y = vy*t+0.5*GRAVITY*t*t;
            if( y <= h )
            {
                //if( DEBUG )	fprintf( stderr, "Ball hits the cliff face\n" );
            }
            else
            {
                // the ball hits the upper deck:
                // the time solution for this is a quadratic equation of the form:
                // at^2 + bt + c = 0.
                // where 'a' multiplies time^2
                //       'b' multiplies time
                //       'c' is a constant
                float a = 0.5*GRAVITY;
                float b = vy;
                float c = -h;
                float disc = b*b - 4.f*a*c;	// quadratic formula discriminant

                // ball doesn't go as high as the upper deck:
                // this should "never happen" ... :-)
                if( disc < 0. )
                {
                    //if( DEBUG )	fprintf( stderr, "Ball doesn't reach the upper deck.\n" );
                    //exit( 1 );	// something is wrong...
                }

                // successfully hits the ground above the cliff:
                // get the intersection:
                disc = sqrtf( disc );
                float t1 = (-b + disc ) / ( 2.f*a );	// time to intersect high ground
                float t2 = (-b - disc ) / ( 2.f*a );	// time to intersect high ground

                // only care about the second intersection
                float tmax = t1;
                if( t2 > t1 )
                    tmax = t2;

                // how far does the ball land horizontlly from the edge of the cliff?
                float upperDist = vx * tmax  -  g;

                // see if the ball hits the castle:
                if(  fabs( upperDist - d ) > TOL )
                {
                    //if( DEBUG )  fprintf( stderr, "Misses the castle at upperDist = %8.3f\n", upperDist );
                }
                else
                {
                    //if( DEBUG )  fprintf( stderr, "Hits the castle at upperDist = %8.3f\n", upperDist );
                    numHits+=1;
                }
            } // if ball clears the cliff face
        }

        dhits[gid] = numHits;
}

int
main( int argc, char* argv[ ] )
{
        TimeOfDaySeed( );

        //int dev = findCudaDevice(argc, (const char **)argv);

        // better to define these here so that the rand() calls don't get into the thread timing:
        float *hvs   = new float [NUMTRIALS];
        float *hths  = new float [NUMTRIALS];
        float *hgs   = new float [NUMTRIALS];
        float *hhs   = new float [NUMTRIALS];
        float *hds   = new float [NUMTRIALS];
        int   *hhits = new int   [NUMTRIALS];

        // fill the random-value arrays:

        for( int n = 0; n < NUMTRIALS; n++ )
        {
            hvs[n]  = Ranf(  VMIN,  VMAX );
            hths[n] = Ranf( THMIN, THMAX );
            hgs[n]  = Ranf(  GMIN,  GMAX );
            hhs[n]  = Ranf(  HMIN,  HMAX );
            hds[n]  = Ranf(  DMIN,  DMAX );
        }


        // allocate device memory:
        float *dvs, *dths, *dgs, *dhs, *dds;
        int   *dhits;

        hipMalloc( &dvs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dths,  NUMTRIALS*sizeof(float) );
        hipMalloc( &dgs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dhs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dds,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dhits, NUMTRIALS*sizeof(int) );
        CudaCheckError();

        // copy host memory to the device:
        hipMemcpy( dvs,  hvs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dths, hths, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dgs,  hgs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dhs,  hhs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dds,  hds,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        CudaCheckError();

        // setup the execution parameters:
        //dim3 grid( NUMBLOCKS, 1, 1 );
        dim3 grid( NUMTRIALS/BLOCKSIZE, 1, 1 );
        dim3 threads( BLOCKSIZE, 1, 1 );

        // allocate cuda events that we'll use for timing:
        hipEvent_t start, stop;
        hipEventCreate( &start );
        hipEventCreate( &stop  );
        CudaCheckError( );

        // let the gpu go quiet:
        hipDeviceSynchronize( );

        // record the start event:
        hipEventRecord( start, NULL );
        CudaCheckError( );

        // execute the kernel:
        MonteCarlo<<< grid, threads >>>( IN dvs, IN dths, IN dgs, IN dhs, IN dds,   OUT dhits );

        // record the stop event:
        hipEventRecord( stop, NULL );
        CudaCheckError( );

        // wait for the stop event to complete:
        hipDeviceSynchronize( );
        hipEventSynchronize( stop );
        CudaCheckError( );

        float msecTotal = 0.0f;
        hipEventElapsedTime( &msecTotal, start, stop );
        CudaCheckError( );

        // compute and print the performance

        double Total_in_Second = 0.001 * (double)msecTotal;
        double megaTrialsPerSecond = (double)NUMTRIALS / ( Total_in_Second ) / 1000000.;
        fprintf(stdout, "%2d, %2d, %6.2lf, ", NUMTRIALS,BLOCKSIZE, megaTrialsPerSecond );


        // copy result from the device to the host:
        hipMemcpy( hhits, dhits, NUMTRIALS*sizeof(int), hipMemcpyDeviceToHost );
        CudaCheckError( );

        // add up the hhits[ ] array: :

        int total_hits = 0;
        for(int i = 0; i < NUMTRIALS; i++){
            total_hits += hhits[i];
        }

        // compute and print the probability:
        float probability = 100*(float)total_hits/(float)( NUMTRIALS );
        fprintf(stdout,"%6.2f%%\n", probability);

        // clean up host memory:
        delete [ ] hvs;
        delete [ ] hths;
        delete [ ] hgs;
        delete [ ] hhs;
        delete [ ] hds;
        delete [ ] hhits;

        // clean up device memory:
        hipFree( dvs );
        hipFree( dths );
        hipFree( dgs );
        hipFree( dhs );
        hipFree( dds );
        hipFree( dhits );
        CudaCheckError( );

	return 0;
}

void
CudaCheckError()
{
        hipError_t e = hipGetLastError( );
        if( e != hipSuccess )
        {
            fprintf( stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e) );
        }
}

float
Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int
Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}

// call this if you want to force your program to use
// a different random number sequence every time you run it:
void
TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}